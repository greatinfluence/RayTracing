#include "hip/hip_runtime.h"
#include "Raytracer.h"

#include "Geometryrepository.h"

__device__ glm::vec3 Raytracing::RayTracing(Ray ray, size_t* root, glm::vec3 m_Background,
	hiprandState& state, int lev, glm::vec3 coef) {
	float dist = floatmax;
	Geometry* hitted = nullptr;
	if (fabs(ray.GetDir().x) > 1.5f) {
		printf("What?\n");
	}
	printf("Raytracing: get ray (%f, %f, %f) -> (%f, %f, %f)\n", ray.GetPos().x, ray.GetPos().y, ray.GetPos().z,
		ray.GetDir().x, ray.GetDir().y, ray.GetDir().z);
	printf("Begin hit check\n");
	static_cast<Cuboid*>(Geometryrepository::GetGeo(*root))->TestHit(ray, dist, hitted);
	printf("End hit check\n");
	if (hitted != nullptr) {
		glm::vec3 hitpos = ray.GetPos() + ray.GetDir() * dist;
		glm::vec3 att, wi, norm = hitted->GetNorm(hitpos);
		Material* mat = hitted->GetMaterial();
		float poss = mat->scatter(hitpos, -ray.GetDir(),
			norm, att, wi, &state);
		if (fabs(dist) > 1e3 || fabs(ray.GetDir().x) > 1.5f) {
			// Impossible!
			printf("What?\n");
		}
		if(lev > 4) {
			float rr = glm::clamp(fmax(coef.r, fmax(coef.g, coef.b)), 0.0f, 0.95f);
			if (lev > 30 || GPURandom::Rand(1.0f, state) > rr) return coef * mat->GetGlow();	
			coef = coef / rr;
		}
		return coef * mat->GetGlow() +
			RayTracing(Ray(hitpos + wi * 1e-4f, wi), root, m_Background, state, lev + 1, coef * att * fabs(glm::dot(wi, norm)) / poss);
	}
	else return coef * m_Background;
}


