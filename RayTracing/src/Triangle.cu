#include "hip/hip_runtime.h"
#include "Triangle.h"

#include <iostream>

__host__ __device__ Triangle::Triangle(la::vec3 v1, la::vec3 v2, la::vec3 v3, la::vec3 norm)
	:m_Vertices{v1, v2, v3}, m_Norm(norm) {
	if (la::l2Norm(norm) < 1e-8) {
		m_Norm = ComputeTriangNorm(v1, v2, v3);
	}
}

la::vec3 Triangle::GetPos(size_t ind) const {
	if (ind >= 3) {
		printf("Triangle::getpos Out of range\n");
	}
	return *(m_Vertices + ind);
}

bool Triangle::OnTriangle(la::vec3 pos) const
{
	auto ed1 = m_Vertices[1] - m_Vertices[0], ed2 = m_Vertices[2] - m_Vertices[0];
	auto vec = pos - m_Vertices[0];
	auto fsthf = la::cross(ed1, vec), sechf = la::cross(vec, ed2);
	if (fsthf.x * sechf.x < -1e-6 || fsthf.y * sechf.y < -1e-6 || fsthf.z * sechf.z < -1e-6)
		return false;
	ed1 = -ed1;
	ed2 = m_Vertices[2] - m_Vertices[1];
	vec = pos - m_Vertices[1];
	fsthf = la::cross(ed1, vec), sechf = la::cross(vec, ed2);
	if (fsthf.x * sechf.x < -1e-6 || fsthf.y * sechf.y < -1e-6 || fsthf.z * sechf.z < -1e-6)
		return false;
	return true;
}
